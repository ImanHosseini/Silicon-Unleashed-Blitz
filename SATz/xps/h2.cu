#include "hip/hip_runtime.h"
#include <cstdlib>
#include <chrono>
#include <cstdio>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

/*
    only god (and my adviser) can judge me, sorry Scott Meyers!
    bible doesnt say anything about not scoping into std.
*/
using namespace std;

#define TC_S 1
#define TC_M 2

#ifndef TC
#define TC TC_S
#endif

#if TC == TC_S
#define TNAME "single"
#endif

#if TC == TC_M
#define TNAME "multi"
#endif

const char *tname = TNAME;

#ifndef VN
#define VN 59790
#endif

#ifndef UR
#define UR 32
#endif

#define TIMES 5

// An experiment with cuRAND

// f0: base version
__global__ void f0(bool *arr, int tcnt, uint64_t seed, int s)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= tcnt) return;
    hiprandState *randS = (hiprandState *)malloc(sizeof(hiprandState));
    hiprand_init(idx*seed, 0, 0, randS);
    for(int i=0; i<s; i++){
        bool rnd = hiprand(randS)%2;
        arr[idx*s+ i] = rnd;
    }
    free(randS);
}



// f0 + Loop Unrolling [S=64 won here]
template <int S>
__global__ void f1(bool *arr, int tcnt, uint64_t seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= tcnt) return;
    hiprandState *randS = (hiprandState *)malloc(sizeof(hiprandState));
    hiprand_init(idx*seed, 0, 0, randS);
    #pragma unroll (S)
    for(int i=0; i<S; i++){
        bool rnd = hiprand(randS)%2;
        arr[idx*S+ i] = rnd;
    }
    free(randS);
}

int main()
{
    for (auto i = 0; i < TIMES; i++)
    {
        printf("[UR: %d | VN: %d]\n",UR, VN);
        int Vcnt = (VN+UR-1)/UR;
        int bsz = (511+Vcnt)/512;
        bool* d_arr;
        // padded to get rid of remainder loop
        hipMalloc(&d_arr, sizeof(bool)*VN);
        // look, i know memory calls sync and this is not needed.
        hipDeviceSynchronize();
        auto t0 = chrono::steady_clock::now();
        // f1<UR><<<bsz,512>>>(d_arr,Vcnt,(uint64_t)time(0));
        f0<<<bsz,512>>>(d_arr,Vcnt,(uint64_t)time(0),UR);
        hipDeviceSynchronize();
        auto t1 = chrono::steady_clock::now();
        chrono::duration<double, micro> elapsed = t1 - t0;
        printf("[%s] %.17g (us)\n", "time", elapsed.count());
    }
}
